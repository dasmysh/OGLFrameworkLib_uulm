#include "hip/hip_runtime.h"
/**
* @file    CUDAAddNoise.cu
* @author  Sebastian Maisch <sebastian.maisch@googlemail.com>
* @date    2014.04.02
*
* @brief   Implementation of the CUDA grid initialization.
*/

#define GLM_FORCE_CUDA
#define GLM_SWIZZLE
#include "hip/hip_runtime.h"
#include "CUDAAddNoise.h"
#include "cudamain.h"
#include "gpgpu/CUDAGrid.h"

#include <glm/glm.hpp>

#pragma warning (disable : 4505)
#ifndef __HIPCC__
    #define __HIPCC__
#endif
#include <hiprand/hiprand_kernel.h>
#include <>
#include <hip/device_functions.h>
#pragma warning (default : 4505)

namespace cgu {
    namespace gpgpu {

        __constant__ float grad[12][3] = { { 0, 1, 1 }, { 0, 1, -1 }, { 0, -1, 1 }, { 0, -1, -1 },
        { 1, 0, 1 }, { 1, 0, -1 }, { -1, 0, 1 }, { -1, 0, -1 },
        { 1, 1, 0 }, { 1, -1, 0 }, { -1, 1, 0 }, { -1, -1, 0 } };


        CUDAAddNoise::CUDAAddNoise(CUDAGrid* theGrid) :
            grid(theGrid)
        {
        }

        CUDAAddNoise::~CUDAAddNoise()
        {
        }

        __global__ void initKernel(glm::uvec3 dim, void* grid, unsigned long long seed,
            float resolution, float strength, glm::vec4 color);

        void CUDAAddNoise::RunKernel(const glm::vec4& color, float resolution, float strength, unsigned int seed) const
        {
            dim3 blocks(static_cast<unsigned int>(grid->GetDimensions().y),
                static_cast<unsigned int>(grid->GetDimensions().z));
            dim3 threads(static_cast<unsigned int>(grid->GetDimensions().x));
            initKernel<<<blocks, threads >>>(grid->GetDimensions(), grid->GetGridPtr(), seed, resolution, strength, color);
            CudaCheckError();
        }

        /*
        * Efficient simplex indexing functions by Bill Licea-Kane, ATI. Thanks!
        * (This was originally implemented as a 1D texture lookup. Nice to avoid that.)
        */
        __device__ void simplex(const glm::vec3& P, glm::vec3& offset1, glm::vec3& offset2)
        {
            glm::vec3 offset0;

            auto isX = glm::step(glm::vec2(P.y, P.z), glm::vec2(P.x, P.x)); // P.x >= P.y ? 1.0 : 0.0;  P.x >= P.z ? 1.0 : 0.0;
            // Accumulate all P.x >= other channels in offset.x
            offset0 = glm::vec3(isX.x + isX.y, 1.0f - isX);        // Accumulate all P.x <  other channels in offset.yz

            auto v_isY = glm::step(glm::fvec1(P.z), glm::fvec1(P.y));  // P.y >= P.z ? 1.0 : 0.0;
            auto isY = v_isY.x;
            offset0.y += isY;              // Accumulate P.y >= P.z in offset.y
            offset0.z += 1.0 - isY;        // Accumulate P.y <  P.z in offset.z

            // offset0 now contains the unique values 0,1,2 in each channel
            // 2 for the channel greater than other channels
            // 1 for the channel that is less than one but greater than another
            // 0 for the channel less than other channels
            // Equality ties are broken in favor of first x, then y
            // (z always loses ties)

            offset2 = glm::clamp(offset0, 0.0f, 1.0f);
            // offset2 contains 1 in each channel that was 1 or 2
            offset1 = glm::clamp(offset0 - 1.0f, 0.0f, 1.0f);
            // offset1 contains 1 in the single channel that was 1
        }

        // , glm::vec3* grad3
        __global__ void initKernel(glm::uvec3 dim, void* grid, unsigned long long seed,
            float resolution, float strength, glm::vec4 color)
        {
            const float F3 = 0.333333333333;
            const float G3 = 0.166666666667;

            // const float ONE = 1.0f / blockDim.x;
            // const float ONEHALF = 0.5f / blockDim.x;
            // const unsigned int gridDimX = blockDim.x;
            const auto gridDimY = gridDim.x;
            // const unsigned int gridDimZ = gridDim.y;
            const auto gridX = threadIdx.x;
            const auto gridY = blockIdx.x;
            const auto gridZ = blockIdx.y;

            // printf("gridPos: (%d,%d,%d)\n", gridX, gridY, gridZ);
            // printf("gridDim: (%d,%d,%d)\n", gridDimX, gridDimY, gridDimZ);

            glm::vec3 P(static_cast<float>(gridX) / resolution,
                static_cast<float>(gridY) / resolution,
                static_cast<float>(gridZ) / resolution);
            // Skew the (x,y,z) space to determine which cell of 6 simplices we're in
            auto s = (P.x + P.y + P.z) * F3; // Factor for 3D skewing
            auto Pi = glm::floor(P + s);
            auto t = (Pi.x + Pi.y + Pi.z) * G3;
            auto P0 = Pi - t; // Unskew the cell origin back to (x,y,z) space
            // Pi = Pi * ONE + ONEHALF; // Integer part, scaled and offset for texture lookup

            auto Pf0 = P - P0;  // The x,y distances from the cell origin

            // For the 3D case, the simplex shape is a slightly irregular tetrahedron.
            // To find out which of the six possible tetrahedra we're in, we need to
            // determine the magnitude ordering of x, y and z components of Pf0.
            glm::vec3 o1;
            glm::vec3 o2;
            simplex(Pf0, o1, o2);

            hiprandStateXORWOW_t rand_state;
            auto uPi = glm::uvec3(__float2uint_rd(Pi.x), __float2uint_rd(Pi.y), __float2uint_rd(Pi.z));
            // glm::uvec3 uo1 = glm::uvec3(__float2uint_rd(o1.x), __float2uint_rd(o1.y), __float2uint_rd(o1.z));
            // glm::uvec3 uo2 = glm::uvec3(__float2uint_rd(o2.x), __float2uint_rd(o2.y), __float2uint_rd(o2.z));
            hiprand_init(seed + uPi.x, uPi.y, uPi.z, &rand_state);
            auto perm0 = hiprand_uniform(&rand_state);
            hiprand_init(seed + uPi.x + o1.x, uPi.y + o1.y, uPi.z + o1.z, &rand_state);
            auto perm1 = hiprand_uniform(&rand_state);
            hiprand_init(seed + uPi.x + o2.x, uPi.y + o2.y, uPi.z + o2.z, &rand_state);
            auto perm2 = hiprand_uniform(&rand_state);
            hiprand_init(seed + uPi.x + 1, uPi.y + 1, uPi.z + 1, &rand_state);
            auto perm3 = hiprand_uniform(&rand_state);

            // Noise contribution from simplex origin
            // float perm0 = 1texture2D(gradTexture, Pi.xy).a;
            // glm::vec3  grad0 = 2texture2D(gradTexture, glm::vec2(perm0, Pi.z)).rgb * 4.0 - 2.0;
            auto pgrad0 = grad[static_cast<unsigned int>(12.0f * perm0)];
            glm::vec3 grad0(pgrad0[0], pgrad0[1], pgrad0[2]);
            float t0 = 0.6 - glm::dot(Pf0, Pf0);
            float n0;
            if (t0 < 0.0) n0 = 0.0;
            else {
                t0 *= t0;
                n0 = t0 * t0 * glm::dot(grad0, Pf0);
            }

            // Noise contribution from second corner
            auto Pf1 = Pf0 - o1 + G3;
            // float perm1 = 3texture2D(gradTexture, Pi.xy + o1.xy*ONE).a;
            // glm::vec3  grad1 = 4texture2D(gradTexture, glm::vec2(perm1, Pi.z + o1.z*ONE)).rgb * 4.0 - 2.0;
            auto pgrad1 = grad[static_cast<unsigned int>(12.0f * perm1)];
            glm::vec3 grad1(pgrad1[0], pgrad1[1], pgrad1[2]);
            float t1 = 0.6 - glm::dot(Pf1, Pf1);
            float n1;
            if (t1 < 0.0) n1 = 0.0;
            else {
                t1 *= t1;
                n1 = t1 * t1 * glm::dot(grad1, Pf1);
            }

            // Noise contribution from third corner
            auto Pf2 = Pf0 - o2 + 2.0f * G3;
            // float perm2 = 5texture2D(gradTexture, Pi.xy + o2.xy*ONE).a;
            // glm::vec3  grad2 = 6texture2D(gradTexture, glm::vec2(perm2, Pi.z + o2.z*ONE)).rgb * 4.0 - 2.0;
            auto pgrad2 = grad[static_cast<unsigned int>(12.0f * perm2)];
            glm::vec3 grad2(pgrad2[0], pgrad2[1], pgrad2[2]);
            float t2 = 0.6 - glm::dot(Pf2, Pf2);
            float n2;
            if (t2 < 0.0) n2 = 0.0;
            else {
                t2 *= t2;
                n2 = t2 * t2 * glm::dot(grad2, Pf2);
            }

            // Noise contribution from last corner
            auto Pf3 = Pf0 - glm::vec3(1.0 - 3.0*G3);
            // float perm3 = 7texture2D(gradTexture, Pi.xy + glm::vec2(ONE, ONE)).a;
            // glm::vec3  grad3 = 8texture2D(gradTexture, glm::vec2(perm3, Pi.z + ONE)).rgb * 4.0 - 2.0;
            auto pgrad3 = grad[static_cast<unsigned int>(12.0f * perm3)];
            glm::vec3 grad3(pgrad3[0], pgrad3[1], pgrad3[2]);
            float t3 = 0.6 - glm::dot(Pf3, Pf3);
            float n3;
            if (t3 < 0.0) n3 = 0.0;
            else {
                t3 *= t3;
                n3 = t3 * t3 * glm::dot(grad3, Pf3);
            }

            auto gridPtr = static_cast<char*>(grid);
            auto pitch = dim.x * sizeof(glm::vec4);
            auto slicePitch = pitch * gridDimY;

            auto slicePtr = gridPtr + (gridZ * slicePitch);
            auto linePtr = reinterpret_cast<glm::vec4*>(slicePtr + (gridY * pitch));
            // glm::vec4* gridPos = &gridPtr[(gridZ * slicePitch) + (gridY * pitch) + gridX];
            auto gridPos = &linePtr[gridX];
            // scale result to [0,1] and adjust to strength and color
            // gridPtr[(gridZ * slicePitch) + (gridY * pitch) + gridX] =
            //     color * strength * ((16.0f * (n0 + n1 + n2 + n3)) + 0.8f);
            *gridPos += color * strength * ((16.0f * (n0 + n1 + n2 + n3)) + 0.5f);

            // *gridPos = glm::vec4(1.0f, 0.0f, 0.0f, 1.0f);
        }
    }
}