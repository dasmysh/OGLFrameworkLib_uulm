/**
* @file    CUDAGrid.cpp
* @author  Sebastian Maisch <sebastian.maisch@googlemail.com>
* @date    2014.04.02
*
* @brief   Implementation of the CUDA grid.
*/

#include "CUDAGrid.h"
#include "cudamain.h"
#include "gpgpu/CUDAImage.h"

#include <glm/glm.hpp>

namespace cgu {
    namespace gpgpu {

        /// <summary>
        /// Initializes a new instance of the <see cref="CUDAGrid"/> class.
        /// </summary>
        /// <param name="width">The width of the grid.</param>
        /// <param name="height">The height of the grid.</param>
        /// <param name="depth">The depth of the grid.</param>
        /// <param name="elementSize">Size of one element.</param>
        CUDAGrid::CUDAGrid(unsigned int width, unsigned int height, unsigned int depth, size_t elementSize) :
            size(make_hipExtent(width * elementSize, height, depth)),
            dimensions(width, height, depth)
        {
            assert(width > 0);
            assert(height > 0);
            assert(depth > 0);
            CudaSafeCall(hipMalloc(&grid, size.width * size.height * size.depth));
        }

        /// <summary>
        /// Finalizes an instance of the <see cref="CUDAGrid"/> class.
        /// </summary>
        CUDAGrid::~CUDAGrid()
        {
            CudaSafeCall(hipFree(grid));
        }

        /// <summary>
        /// Initializes the grid.
        /// </summary>
        /// <param name="value">The value to initialize each field.</param>
        void CUDAGrid::InitGrid(int value) const
        {
            CudaSafeCall(hipMemset(grid, value, size.width * size.height * size.depth));
        }

        /// <summary>
        /// Initializes the grid using a texture. Will only work for 2d grids.
        /// </summary>
        /// <param name="texture">The texture.</param>
        void CUDAGrid::InitGrid(CUDAImage* texture) const
        {
            assert(size.depth == 1);
            auto src = texture->GetMappedArray();
            CudaSafeCall(hipMemcpy2DFromArray(grid, size.width, src, 0, 0, size.width,
                size.height, hipMemcpyDeviceToDevice));
        }

        /// <summary>
        /// Copies the grids content to texture. Works only with 2d grids.
        /// </summary>
        /// <param name="texture">The texture.</param>
        void CUDAGrid::CopyToTexture(CUDAImage* texture) const
        {
            assert(size.depth == 1);
            auto dst = texture->GetMappedArray();
            CudaSafeCall(hipMemcpyToArray(dst, 0, 0, grid, size.width * size.height,
                hipMemcpyDeviceToDevice));
        }


        /// <summary>
        /// Copies grid contents to host.
        /// </summary>
        /// <param name="ptr">The pointer to allocated host memory.</param>
        /// <param name="destSize">Size of the host memory in elements.</param>
        /// <param name="elementSize">Size of a single element.</param>
        void CUDAGrid::CopyToHost(void* ptr, const glm::uvec3 destSize, size_t elementSize) const
        {
            assert(dimensions.x == destSize.x);
            assert(dimensions.y == destSize.y);
            assert(dimensions.z == destSize.z);
            assert(size.width == (destSize.x * elementSize));
            CudaSafeCall(hipMemcpy(ptr, grid, size.width * size.height * size.depth,
                hipMemcpyDeviceToHost));
        }
    }
}
